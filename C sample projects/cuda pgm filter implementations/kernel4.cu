#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include "kernels.h"
#include <stdio.h>
#include <assert.h>
#include <time.h>
#define COMPUTE_UNIT (2*36)
#define THREAD_AMOUNT (256)
#define WARPTHREADS (1024)

__device__ int32_t apply2d4(const int8_t *f, int32_t dimension, const int32_t *original, int32_t width, int32_t height, int row, int column) {
    int size = width * height;
    int32_t pixel = 0;
    row -= dimension/2;
    column-= dimension/2;
    // loop to add all the pixels 
    for(int i = 0;i<dimension;i++){
        for (int j = 0;j<dimension;j++){
            int location = row*width + column;
            if (0 <= location && location < size && row > -1 && column > -1 && row < height && column < width){
                pixel += original[location] * f[i * dimension + j];
            }
            column++;
            
        }
        row++;
        column-=dimension;
    }
    
    return pixel;

}
void print_run4(float time_cpu, int kernel, float time_gpu_computation, float time_gpu_transfer_in, float time_gpu_transfer_out) {
    printf("%12.6f ", time_cpu);
    printf("%5d ", kernel);
    printf("%12.6f ", time_gpu_computation);
    printf("%14.6f ", time_gpu_transfer_in);
    printf("%15.6f ", time_gpu_transfer_out);
    printf("%13.2f ", time_cpu / time_gpu_computation);
    printf("%7.2f\n", time_cpu / (time_gpu_computation + time_gpu_transfer_in +
                            time_gpu_transfer_out));
}

__global__ void warpReduceMin4(int32_t *val, int N, int32_t *output)
{
    int base = (blockIdx.x * blockDim.x);
    int id = (threadIdx.x) ;
    int localn = 4;
    if (blockDim.x == blockIdx.x-1){
        localn = N - base;
    }
    int n = localn;
    localn /= 2;
    while (localn > 0){
        if(id < localn){
            atomicMin(&val[base + id], val[base + id + localn]);
        }
        localn /= 2;
        __syncthreads();
    }
    if (threadIdx.x == 0){
        atomicMin(&val[base + id], val[base + n-1]);
        output[blockIdx.x] = val[base + id];
    }
    __syncthreads();
}

__global__ void warpReduceMax4(int32_t *val, int N, int32_t *output)
{
    int base = (blockIdx.x * blockDim.x);
    int id = (threadIdx.x) ;
    int localn = 4;
    if (gridDim.x-1 == blockIdx.x){
        localn = N - base;
    }
    int n = localn;
    localn /= 2;
    while (localn > 0 ){
        if(id < localn){
            atomicMax(&val[base + id], val[base + id + localn]);
        }
        localn /= 2;
        __syncthreads();
    }
    if (threadIdx.x == 0){
        atomicMax(&val[base + id], val[base + n-1]);
        output[blockIdx.x] = val[base + id];
    }
    __syncthreads();
}

int32_t findmax4(int N, int32_t *gpumem){
    int blocks = N/WARPTHREADS;
    blocks++;
    if(blocks>1){
        int32_t *max_out;
        hipMalloc(&max_out, N*sizeof(int32_t));
        warpReduceMax4<<<blocks,WARPTHREADS>>>(gpumem, N, max_out);
        int32_t m = findmax4(blocks, max_out);
        hipFree(max_out);
        return m;
    }
    else{
        int32_t m = -2147483647;
        int32_t *hostmem;
        hostmem = (int32_t*)malloc(sizeof(int32_t) * N);
        hipMemcpy(hostmem, gpumem, sizeof(int32_t) * N,  hipMemcpyDeviceToHost);
        for(int i = 0;i<N;i++){
            if(hostmem[i] > m){
                m = hostmem[i];
            }
        }
        free(hostmem);
        return m;
    }
    return -1;
}

int32_t findmin4(int N, int32_t *gpumem){
    int blocks = N/WARPTHREADS;
    blocks++;
    if(blocks>1){
        int32_t *min_out;
        hipMalloc(&min_out, N*sizeof(int32_t));
        warpReduceMin4<<<blocks,WARPTHREADS>>>(gpumem, N, min_out);
        int32_t m = findmin4(blocks, min_out);
        hipFree(min_out);
        return m;
    }
    else{
        int32_t m = 2147483647;
        int32_t *hostmem;
        hostmem = (int32_t*)malloc(sizeof(int32_t) * N);
        hipMemcpy(hostmem, gpumem, sizeof(int32_t) * N,  hipMemcpyDeviceToHost);
        for(int i = 0;i<N;i++){
            if(hostmem[i] < m){
                m = hostmem[i];
            }
        }
        free(hostmem);
        return m;
    }
    return -1;
}


void run_kernel4(const int8_t *filter, int32_t dimension, const int32_t *input,
                 int32_t *output, int32_t width, int32_t height, double cputime) 
{
    // init
    int32_t *gpu_in;
    int32_t *gpu_out;
    int8_t *gpu_filter;
    int32_t *buf_min;
    int32_t *buf_max;
    float transfer_in, computation_time, transfer_out; 
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // malloc 
    hipMalloc(&gpu_filter, dimension*dimension*sizeof(int8_t));
    hipMalloc(&gpu_in, width*height*sizeof(int32_t));
    hipMalloc(&gpu_out, width*height*sizeof(int32_t));
    hipMalloc(&buf_min, width*height*sizeof(int32_t));
    hipMalloc(&buf_max, width*height*sizeof(int32_t));
    hipMemcpy(gpu_in, input, width*height*sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_filter, filter, dimension*dimension*sizeof(int8_t), hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&transfer_in, start, stop);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // run kernel 
    kernel4<<<COMPUTE_UNIT,THREAD_AMOUNT>>>(gpu_filter, dimension, gpu_in, gpu_out, width, height); 
    
    hipMemcpy(buf_max, gpu_out, width*height*sizeof(int32_t), hipMemcpyDeviceToDevice);
    hipMemcpy(buf_min, gpu_out, width*height*sizeof(int32_t), hipMemcpyDeviceToDevice);
    // runs with atmoic instructions 
    int warplargest = findmax4(width*height, buf_max);
    int warpsmallest = findmin4(width*height, buf_min);

    if (warpsmallest != warplargest){
        normalize4<<<COMPUTE_UNIT,THREAD_AMOUNT>>>(gpu_out, width, height, warpsmallest, warplargest);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&computation_time, start, stop);


    hipEventRecord(start);

    hipMemcpy(output, gpu_out, width*height*sizeof(int32_t), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&transfer_out, start, stop);
    print_run4(cputime, 4, computation_time, transfer_in, transfer_out);
    // cleanup 
    hipFree(gpu_in);
    hipFree(gpu_out);
    hipFree(gpu_filter);
    hipFree(buf_max);
    hipFree(buf_min);
    hipEventDestroy(start);
    hipEventDestroy(stop);

}

__global__ void kernel4(const int8_t *filter, int32_t dimension,
                        const int32_t *input, int32_t *output, int32_t width,
                        int32_t height) 
{
    int id = (threadIdx.x  + (blockIdx.x * blockDim.x) ) ;
    // int workload = (width * height) / (COMPUTE_UNIT * blockDim.x);
    int workload =  (COMPUTE_UNIT * blockDim.x);
    int location = id;
    while (location < width * height){
        output[location] = apply2d4(filter, dimension, input, width, height, location/width, location%width);
        location += workload;
    }
    __syncthreads();
}

__global__ void normalize4(int32_t *image, int32_t width, int32_t height,
                           int32_t smallest, int32_t biggest) 
{
    int id = (threadIdx.x  + (blockIdx.x * blockDim.x) ) ;
    // int workload = (width * height) / (COMPUTE_UNIT * blockDim.x);
    int workload =  (COMPUTE_UNIT * blockDim.x);
    int location = id;
    while (location < width * height){
        image[location] = ((image[location] - smallest) * 255) / (biggest - smallest);
        location += workload;
    }
    __syncthreads();
}
