#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include "kernels.h"
#include <stdio.h>
#include <assert.h>
#include <time.h>
#define COMPUTE_UNIT (36*2)
#define THREAD_AMOUNT (256)
#define WARPTHREADS (1024)

__global__ void warpReduceMin3(int32_t *val, int N, int32_t *output)
{
    int base = (blockIdx.x * blockDim.x);
    int id = (threadIdx.x) ;
    int localn = 4;
    if (blockDim.x == blockIdx.x-1){
        localn = N - base;
    }
    int n = localn;
    localn /= 2;
    while (localn > 0){
        if(id < localn){
            atomicMin(&val[base + id], val[base + id + localn]);
        }
        localn /= 2;
        __syncthreads();
    }
    if (threadIdx.x == 0){
        atomicMin(&val[base + id], val[base + n-1]);
        output[blockIdx.x] = val[base + id];
    }
    __syncthreads();
}

__global__ void warpReduceMax3(int32_t *val, int N, int32_t *output)
{
    int base = (blockIdx.x * blockDim.x);
    int id = (threadIdx.x) ;
    int localn = 4;
    if (gridDim.x-1 == blockIdx.x){
        localn = N - base;
    }
    int n = localn;
    localn /= 2;
    while (localn > 0 ){
        if(id < localn){
            atomicMax(&val[base + id], val[base + id + localn]);
        }
        localn /= 2;
        __syncthreads();
    }
    if (threadIdx.x == 0){
        atomicMax(&val[base + id], val[base + n-1]);
        output[blockIdx.x] = val[base + id];
    }
    __syncthreads();
}



int32_t findmax3(int N, int32_t *gpumem){
    int blocks = N/WARPTHREADS;
    blocks++;
    if(blocks>1){
        int32_t *max_out;
        hipMalloc(&max_out, N*sizeof(int32_t));
        warpReduceMax3<<<blocks,WARPTHREADS>>>(gpumem, N, max_out);
        int32_t m = findmax3(blocks, max_out);
        hipFree(max_out);
        return m;
    }
    else{
        int32_t m = -2147483647;
        int32_t *hostmem;
        hostmem = (int32_t*)malloc(sizeof(int32_t) * N);
        hipMemcpy(hostmem, gpumem, sizeof(int32_t) * N,  hipMemcpyDeviceToHost);
        for(int i = 0;i<N;i++){
            if(hostmem[i] > m){
                m = hostmem[i];
            }
        }
        free(hostmem);
        return m;
    }
    return -1;
}

int32_t findmin3(int N, int32_t *gpumem){
    int blocks = N/WARPTHREADS;
    blocks++;
    if(blocks>1){
        int32_t *min_out;
        hipMalloc(&min_out, N*sizeof(int32_t));
        warpReduceMin3<<<blocks,WARPTHREADS>>>(gpumem, N, min_out);
        int32_t m = findmin3(blocks, min_out);
        hipFree(min_out);
        return m;
    }
    else{
        int32_t m = 2147483647;
        int32_t *hostmem;
        hostmem = (int32_t*)malloc(sizeof(int32_t) * N);
        hipMemcpy(hostmem, gpumem, sizeof(int) * N,  hipMemcpyDeviceToHost);
        for(int i = 0;i<N;i++){
            if(hostmem[i] < m){
                m = hostmem[i];
            }
        }
        free(hostmem);
        return m;
    }
    return -1;
}


__device__ int32_t apply2d3(const int8_t *f, int32_t dimension, const int32_t *original, int32_t width, int32_t height, int row, int column) {
    int size = width * height;
    int32_t pixel = 0;
    row -= dimension/2;
    column-= dimension/2;
    // loop to add all the pixels 
    for(int i = 0;i<dimension;i++){
        for (int j = 0;j<dimension;j++){
            int location = row*width + column;
            if (0 <= location && location < size && row > -1 && column > -1 && row < height && column < width){
                pixel += original[location] * f[i * dimension + j];
            }
            column++;
            
        }
        row++;
        column-=dimension;
    }
    
    return pixel;

}

void print_run3(float time_cpu, int kernel, float time_gpu_computation, float time_gpu_transfer_in, float time_gpu_transfer_out) {
    printf("%12.6f ", time_cpu);
    printf("%5d ", kernel);
    printf("%12.6f ", time_gpu_computation);
    printf("%14.6f ", time_gpu_transfer_in);
    printf("%15.6f ", time_gpu_transfer_out);
    printf("%13.2f ", time_cpu / time_gpu_computation);
    printf("%7.2f\n", time_cpu / (time_gpu_computation + time_gpu_transfer_in +
                            time_gpu_transfer_out));
}


void run_kernel3(const int8_t *filter, int32_t dimension, const int32_t *input,
                 int32_t *output, int32_t width, int32_t height, double cputime) 
{
    // init
    int32_t *gpu_in;
    int32_t *gpu_out;
    int8_t *gpu_filter;
    int32_t *buf_min;
    int32_t *buf_max;
    float transfer_in, computation_time, transfer_out; 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // malloc 
    hipMalloc(&gpu_filter, dimension*dimension*sizeof(int8_t));
    hipMalloc(&gpu_in, width*height*sizeof(int32_t));
    hipMalloc(&gpu_out, width*height*sizeof(int32_t));
    hipMalloc(&buf_min, width*height*sizeof(int32_t));
    hipMalloc(&buf_max, width*height*sizeof(int32_t));
    hipMemcpy(gpu_in, input, width*height*sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(gpu_filter, filter, dimension*dimension*sizeof(int8_t), hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&transfer_in, start, stop);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // run kernel 
    kernel3<<<COMPUTE_UNIT,THREAD_AMOUNT>>>(gpu_filter, dimension, gpu_in, gpu_out, width, height); 

    hipMemcpy(buf_max, gpu_out, width*height*sizeof(int32_t), hipMemcpyDeviceToDevice);
    hipMemcpy(buf_min, gpu_out, width*height*sizeof(int32_t), hipMemcpyDeviceToDevice);
    // runs with atmoic instructions 
    int warplargest = findmax3(width*height, buf_max);
    int warpsmallest = findmin3(width*height, buf_min);

    if (warpsmallest != warplargest){
        normalize3<<<COMPUTE_UNIT,THREAD_AMOUNT>>>(gpu_out, width, height, warpsmallest, warplargest);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&computation_time, start, stop);
    hipEventRecord(start);
    
    hipMemcpy(output, gpu_out, width*height*sizeof(int32_t), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&transfer_out, start, stop);
    print_run3(cputime, 3, computation_time, transfer_in, transfer_out);
    hipFree(gpu_in);
    hipFree(gpu_out);
    hipFree(gpu_filter);
    hipFree(buf_max);
    hipFree(buf_min);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    
}

__global__ void kernel3(const int8_t *filter, int32_t dimension,
                        const int32_t *input, int32_t *output, int32_t width,
                        int32_t height) 
                        
{
    int id = (threadIdx.x  + (blockIdx.x * blockDim.x) ) ;
    int workload = (width * height) / (COMPUTE_UNIT * blockDim.x);
    int location = workload *  id;
    int end = workload * (id+1);
    if (blockIdx.x == COMPUTE_UNIT -1 && threadIdx.x == THREAD_AMOUNT-1){
        end = (width * height);
    }
    
    while (location < width * height && location < end){
        output[location] = apply2d3(filter, dimension, input, width, height, location/width, location%width);
        location++;
    }
    
    __syncthreads();

}


__global__ void normalize3(int32_t *image, int32_t width, int32_t height,
                           int32_t smallest, int32_t biggest) 
{
    int id = (threadIdx.x  + (blockIdx.x * blockDim.x) ) ;
    int workload = (width * height) / (COMPUTE_UNIT * blockDim.x);
    int location = workload * id;
    int end = workload * (id+1);
    if (blockIdx.x == COMPUTE_UNIT -1 && threadIdx.x == THREAD_AMOUNT-1){
        end = (width * height);
    }
    while (location < width * height && location < end){
        if (width * height == 1){
            printf("before %d %d %d %d\n", image[location],(image[location] - smallest) * 255, biggest - smallest, (image[location] - smallest) * 255/ biggest - smallest );
        }
        image[location] = ((image[location] - smallest) * 255) / (biggest - smallest);
        if (width * height == 1){
            printf("after %d\n", image[location]);
        }
        location++;
    }
    __syncthreads();
   
}
